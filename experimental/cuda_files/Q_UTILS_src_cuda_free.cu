extern "C" {
//START_INCLUDES
#include "q_incs.h"
//STOP_INCLUDES
#include "_cuda_free.h"
}

//START_FUNC_DECL
int
cuda_free(
    void *ptr
    )
//STOP_FUNC_DECL
{
  int status = 0;
  // CUDA: free memory allocated using hipMallocManaged
  hipFree(ptr);
  return status;
}

