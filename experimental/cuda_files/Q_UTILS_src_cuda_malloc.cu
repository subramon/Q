extern "C" {
//START_INCLUDES
#include "q_incs.h"
//STOP_INCLUDES
#include "_cuda_malloc.h"
}

//START_FUNC_DECL
int
cuda_malloc(
    void **ptr,
    int64_t N
    )
//STOP_FUNC_DECL
{
  int status = 0;
  // CUDA: malloc using hipMallocManaged
  hipMallocManaged(ptr, N);
BYE:
  return status;
}
