#include "hip/hip_runtime.h"
extern "C" {
#include "q_incs.h"
#include "_mmap.h"
}


#define NODE_TYPE int32_t
#define MAXLINE 65535

/*
static __device__ __inline__ uint32_t __mysmid(){
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
  return smid;}

static __device__ __inline__ uint32_t __mywarpid(){
  uint32_t warpid;
  asm volatile("mov.u32 %0, %%warpid;" : "=r"(warpid));
  return warpid;}

static __device__ __inline__ uint32_t __mylaneid(){
  uint32_t laneid;
  asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneid));
  return laneid;}
*/

__global__
static void
any_change(
    NODE_TYPE *lb,
    NODE_TYPE *ub,
    NODE_TYPE *to,
    NODE_TYPE *lbl,
    uint64_t n_nodes,
    bool *is_any_change
    )
{
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t stride = blockDim.x * gridDim.x;
  for (uint64_t i = index; i < n_nodes; i += stride) {
    // printf("I am thread %d, my SM ID is %d, my warp ID is %d, and my warp lane is %d\n", i, __mysmid(), __mywarpid(), __mylaneid());
    bool l_is_any_change = false;
    if ( ub[i] <= lb[i] ) { continue; }
    NODE_TYPE minval = lbl[i];
    for ( int64_t j = lb[i]; j < ub[i]; j++ ) {
      minval = mcr_min(minval, lbl[to[j]]);
    }
    if ( lbl[i] != minval ) {
      l_is_any_change = true;
      lbl[i] = minval;
    }

    if ( ( l_is_any_change ) && ( *is_any_change == false ) ) {
      // printf("Changed the global is_any_change\n");
      *is_any_change = true;
    }
  }
}


int
main(
    int argc,
    char **argv
    )
{
  int status = 0;
  NODE_TYPE *lbl = NULL;
  NODE_TYPE *lb = NULL;
  NODE_TYPE *ub = NULL;
  NODE_TYPE *to = NULL;
  bool *is_any_change = NULL;
  char *lb_X = NULL; size_t lb_nX = 0;
  char *ub_X = NULL; size_t ub_nX = 0;
  char *to_X = NULL; size_t to_nX = 0;

  if ( argc != 1 ) { status = -1; return status; }

  status = rs_mmap("lb.bin", &lb_X, &lb_nX, 0);

  status = rs_mmap("ub.bin", &ub_X, &ub_nX, 0);

  status = rs_mmap("to.bin", &to_X, &to_nX, 0);

  uint64_t n_nodes = lb_nX / sizeof(NODE_TYPE);
  fprintf(stderr, "Working on  %ld nodes \n", n_nodes);

  // Allocate memory for lb, ub, to & lbl using hipMallocManaged
  hipMallocManaged(&lbl, lb_nX);
  hipMallocManaged(&lb, lb_nX);
  hipMallocManaged(&ub, ub_nX);
  hipMallocManaged(&to, to_nX);
  hipMallocManaged(&is_any_change, sizeof(bool));

  if ( lbl == NULL ) { printf("cuda malloc failed for lbl\n"); return -1; }
  if ( lb == NULL ) { printf("cuda malloc failed for lb\n"); return -1; }
  if ( ub == NULL ) { printf("cuda malloc failed for ub\n"); return -1; }
  if ( to == NULL ) { printf("cuda malloc failed for to\n"); return -1; }
  if ( is_any_change == NULL ) { printf("cuda malloc failed for is_any_change\n"); return -1; }

  printf("Memory allocation done\n");

  // Initialize lbl, lb, ub, to, is_any_change
  for ( unsigned int i = 0; i < n_nodes; i++ ) {
    lbl[i] = i;
  }
  memcpy(lb, lb_X, lb_nX);
  memcpy(ub, ub_X, ub_nX);
  memcpy(to, to_X, to_nX);
  *is_any_change = true; // just to get in the first tome

  uint64_t blockSize = 256;
  uint64_t numBlocks = (n_nodes + 256 - 1) / blockSize;

  for ( int iter = 0; *is_any_change == true; iter++ ) {
    // any_change<<<numBlocks, blockSize>>>(lb, ub, to, lbl, n_nodes, &is_any_change);
    for ( int i = 0; i < n_nodes; i++ ) {
      printf("%d\t", lbl[i]);
    }
    printf("\n");
    *is_any_change = false;
    any_change<<<1, 3>>>(lb, ub, to, lbl, n_nodes, is_any_change);
    hipDeviceSynchronize();
    fprintf(stderr, "Pass %d \n", iter);
  }

  hipFree(lbl);
  hipFree(lb);
  hipFree(ub);
  hipFree(to);
  hipFree(is_any_change);
  return status;

}

